
#include <hip/hip_runtime.h>
//
// Copyright (c) 2025 Łukasz Szpakowski
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.
//
#define TILE_WIDTH      32

#define MTHREAD_COUNT   16
#define MTILE_WIDTH     (MTHREAD_COUNT << 2)

extern "C" {
  __global__ void transpose_a(const float *a, float *b, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      b[m * i + j] = a[n * j + i];
    }
  }
  
  __global__ void add_a_b(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] + b[m * i + j];
    }
  }

  __global__ void add_at_b(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] + b[m * i + j];
    }
  }

  __global__ void add_a_bt(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] + b[n * j + i];
    }
  }

  __global__ void add_at_bt(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] + b[n * j + i];
    }
  }

  __global__ void sub_a_b(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] - b[m * i + j];
    }
  }

  __global__ void sub_at_b(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] - b[m * i + j];
    }
  }

  __global__ void sub_a_bt(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] - b[n * j + i];
    }
  }

  __global__ void sub_at_bt(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] - b[n * j + i];
    }
  }

  __global__ void mul_a_b(const float *a, const float *b, float *c, size_t n, size_t m, size_t l)
  {
    __shared__ float as[MTILE_WIDTH][MTHREAD_COUNT];
    __shared__ float bs[MTHREAD_COUNT][MTILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x << 2;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y << 2;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    size_t bi = ti << 2;
    size_t bj = tj << 2;
    float ar1;
    float ar2;
    float ar3;
    float ar4;
    float br1;
    float br2;
    float br3;
    float br4;
    float cr11 = 0.0f;
    float cr12 = 0.0f;
    float cr13 = 0.0f;
    float cr14 = 0.0f;
    float cr21 = 0.0f;
    float cr22 = 0.0f;
    float cr23 = 0.0f;
    float cr24 = 0.0f;
    float cr31 = 0.0f;
    float cr32 = 0.0f;
    float cr33 = 0.0f;
    float cr34 = 0.0f;
    float cr41 = 0.0f;
    float cr42 = 0.0f;
    float cr43 = 0.0f;
    float cr44 = 0.0f;
    for(k = 0; k < l; k += MTHREAD_COUNT) {
      size_t tk;
      as[bi + 0][tj] = 0.0f;
      if(i + 0 < n && k + tj < l) {
        as[bi + 0][tj] = a[l * (i + 0) + k + tj];
      }
      as[bi + 1][tj] = 0.0f;
      if(i + 1 < n && k + tj < l) {
        as[bi + 1][tj] = a[l * (i + 1) + k + tj];
      }
      as[bi + 2][tj] = 0.0f;
      if(i + 2 < n && k + tj < l) {
        as[bi + 2][tj] = a[l * (i + 2) + k + tj];
      }
      as[bi + 3][tj] = 0.0f;
      if(i + 3 < n && k + tj < l) {
        as[bi + 3][tj] = a[l * (i + 3) + k + tj];
      }
      bs[ti][bj + 0] = 0.0f;
      if(j + 0 < m && k + ti < l) {
        bs[ti][bj + 0] = b[m * (k + ti) + j + 0];
      }
      bs[ti][bj + 1] = 0.0f;
      if(j + 1 < m && k + ti < l) {
        bs[ti][bj + 1] = b[m * (k + ti) + j + 1];
      }
      bs[ti][bj + 2] = 0.0f;
      if(j + 2 < m && k + ti < l) {
        bs[ti][bj + 2] = b[m * (k + ti) + j + 2];
      }
      bs[ti][bj + 3] = 0.0f;
      if(j + 3 < m && k + ti < l) {
        bs[ti][bj + 3] = b[m * (k + ti) + j + 3];
      }
      __syncthreads();
      for(tk = 0; tk < MTHREAD_COUNT; tk++) {
        ar1 = as[bi + 0][tk];
        ar2 = as[bi + 1][tk];
        ar3 = as[bi + 2][tk];
        ar4 = as[bi + 3][tk];
        br1 = bs[tk][bj + 0];
        br2 = bs[tk][bj + 1];
        br3 = bs[tk][bj + 2];
        br4 = bs[tk][bj + 3];
        cr11 += ar1 * br1;
        cr12 += ar1 * br2;
        cr13 += ar1 * br3;
        cr14 += ar1 * br4;
        cr21 += ar2 * br1;
        cr22 += ar2 * br2;
        cr23 += ar2 * br3;
        cr24 += ar2 * br4;
        cr31 += ar3 * br1;
        cr32 += ar3 * br2;
        cr33 += ar3 * br3;
        cr34 += ar3 * br4;
        cr41 += ar4 * br1;
        cr42 += ar4 * br2;
        cr43 += ar4 * br3;
        cr44 += ar4 * br4;
      }
      __syncthreads();
    }
    if(i + 0 < n && j + 0 < m) {
      c[m * (i + 0) + j + 0] = cr11;
    }
    if(i + 0 < n && j + 1 < m) {
      c[m * (i + 0) + j + 1] = cr12;
    }
    if(i + 0 < n && j + 2 < m) {
      c[m * (i + 0) + j + 2] = cr13;
    }
    if(i + 0 < n && j + 3 < m) {
      c[m * (i + 0) + j + 3] = cr14;
    }
    if(i + 1 < n && j + 0 < m) {
      c[m * (i + 1) + j + 0] = cr21;
    }
    if(i + 1 < n && j + 1 < m) {
      c[m * (i + 1) + j + 1] = cr22;
    }
    if(i + 1 < n && j + 2 < m) {
      c[m * (i + 1) + j + 2] = cr23;
    }
    if(i + 1 < n && j + 3 < m) {
      c[m * (i + 1) + j + 3] = cr24;
    }
    if(i + 2 < n && j + 0 < m) {
      c[m * (i + 2) + j + 0] = cr31;
    }
    if(i + 2 < n && j + 1 < m) {
      c[m * (i + 2) + j + 1] = cr32;
    }
    if(i + 2 < n && j + 2 < m) {
      c[m * (i + 2) + j + 2] = cr33;
    }
    if(i + 2 < n && j + 3 < m) {
      c[m * (i + 2) + j + 3] = cr34;
    }
    if(i + 3 < n && j + 0 < m) {
      c[m * (i + 3) + j + 0] = cr41;
    }
    if(i + 3 < n && j + 1 < m) {
      c[m * (i + 3) + j + 1] = cr42;
    }
    if(i + 3 < n && j + 2 < m) {
      c[m * (i + 3) + j + 2] = cr43;
    }
    if(i + 3 < n && j + 3 < m) {
      c[m * (i + 3) + j + 3] = cr44;
    }
  }

  __global__ void mul_at_b(const float *a, const float *b, float *c, size_t n, size_t m, size_t l)
  {
    __shared__ float as[MTILE_WIDTH][MTHREAD_COUNT];
    __shared__ float bs[MTHREAD_COUNT][MTILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x << 2;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y << 2;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    size_t bi = ti << 2;
    size_t bj = tj << 2;
    float ar1;
    float ar2;
    float ar3;
    float ar4;
    float br1;
    float br2;
    float br3;
    float br4;
    float cr11 = 0.0f;
    float cr12 = 0.0f;
    float cr13 = 0.0f;
    float cr14 = 0.0f;
    float cr21 = 0.0f;
    float cr22 = 0.0f;
    float cr23 = 0.0f;
    float cr24 = 0.0f;
    float cr31 = 0.0f;
    float cr32 = 0.0f;
    float cr33 = 0.0f;
    float cr34 = 0.0f;
    float cr41 = 0.0f;
    float cr42 = 0.0f;
    float cr43 = 0.0f;
    float cr44 = 0.0f;
    for(k = 0; k < l; k += MTHREAD_COUNT) {
      size_t tk;
      as[bi + 0][tj] = 0.0f;
      if(i + 0 < n && k + tj < l) {
        as[bi + 0][tj] = a[n * (k + tj) + i + 0];
      }
      as[bi + 1][tj] = 0.0f;
      if(i + 1 < n && k + tj < l) {
        as[bi + 1][tj] = a[n * (k + tj) + i + 1];
      }
      as[bi + 2][tj] = 0.0f;
      if(i + 2 < n && k + tj < l) {
        as[bi + 2][tj] = a[n * (k + tj) + i + 2];
      }
      as[bi + 3][tj] = 0.0f;
      if(i + 3 < n && k + tj < l) {
        as[bi + 3][tj] = a[n * (k + tj) + i + 3];
      }
      bs[ti][bj + 0] = 0.0f;
      if(j + 0 < m && k + ti < l) {
        bs[ti][bj + 0] = b[m * (k + ti) + j + 0];
      }
      bs[ti][bj + 1] = 0.0f;
      if(j + 1 < m && k + ti < l) {
        bs[ti][bj + 1] = b[m * (k + ti) + j + 1];
      }
      bs[ti][bj + 2] = 0.0f;
      if(j + 2 < m && k + ti < l) {
        bs[ti][bj + 2] = b[m * (k + ti) + j + 2];
      }
      bs[ti][bj + 3] = 0.0f;
      if(j + 3 < m && k + ti < l) {
        bs[ti][bj + 3] = b[m * (k + ti) + j + 3];
      }
      __syncthreads();
      for(tk = 0; tk < MTHREAD_COUNT; tk++) {
        ar1 = as[bi + 0][tk];
        ar2 = as[bi + 1][tk];
        ar3 = as[bi + 2][tk];
        ar4 = as[bi + 3][tk];
        br1 = bs[tk][bj + 0];
        br2 = bs[tk][bj + 1];
        br3 = bs[tk][bj + 2];
        br4 = bs[tk][bj + 3];
        cr11 += ar1 * br1;
        cr12 += ar1 * br2;
        cr13 += ar1 * br3;
        cr14 += ar1 * br4;
        cr21 += ar2 * br1;
        cr22 += ar2 * br2;
        cr23 += ar2 * br3;
        cr24 += ar2 * br4;
        cr31 += ar3 * br1;
        cr32 += ar3 * br2;
        cr33 += ar3 * br3;
        cr34 += ar3 * br4;
        cr41 += ar4 * br1;
        cr42 += ar4 * br2;
        cr43 += ar4 * br3;
        cr44 += ar4 * br4;
      }
      __syncthreads();
    }
    if(i + 0 < n && j + 0 < m) {
      c[m * (i + 0) + j + 0] = cr11;
    }
    if(i + 0 < n && j + 1 < m) {
      c[m * (i + 0) + j + 1] = cr12;
    }
    if(i + 0 < n && j + 2 < m) {
      c[m * (i + 0) + j + 2] = cr13;
    }
    if(i + 0 < n && j + 3 < m) {
      c[m * (i + 0) + j + 3] = cr14;
    }
    if(i + 1 < n && j + 0 < m) {
      c[m * (i + 1) + j + 0] = cr21;
    }
    if(i + 1 < n && j + 1 < m) {
      c[m * (i + 1) + j + 1] = cr22;
    }
    if(i + 1 < n && j + 2 < m) {
      c[m * (i + 1) + j + 2] = cr23;
    }
    if(i + 1 < n && j + 3 < m) {
      c[m * (i + 1) + j + 3] = cr24;
    }
    if(i + 2 < n && j + 0 < m) {
      c[m * (i + 2) + j + 0] = cr31;
    }
    if(i + 2 < n && j + 1 < m) {
      c[m * (i + 2) + j + 1] = cr32;
    }
    if(i + 2 < n && j + 2 < m) {
      c[m * (i + 2) + j + 2] = cr33;
    }
    if(i + 2 < n && j + 3 < m) {
      c[m * (i + 2) + j + 3] = cr34;
    }
    if(i + 3 < n && j + 0 < m) {
      c[m * (i + 3) + j + 0] = cr41;
    }
    if(i + 3 < n && j + 1 < m) {
      c[m * (i + 3) + j + 1] = cr42;
    }
    if(i + 3 < n && j + 2 < m) {
      c[m * (i + 3) + j + 2] = cr43;
    }
    if(i + 3 < n && j + 3 < m) {
      c[m * (i + 3) + j + 3] = cr44;
    }
  }

  __global__ void mul_a_bt(const float *a, const float *b, float *c, size_t n, size_t m, size_t l)
  {
    __shared__ float as[MTILE_WIDTH][MTHREAD_COUNT];
    __shared__ float bs[MTHREAD_COUNT][MTILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x << 2;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y << 2;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    size_t bi = ti << 2;
    size_t bj = tj << 2;
    float ar1;
    float ar2;
    float ar3;
    float ar4;
    float br1;
    float br2;
    float br3;
    float br4;
    float cr11 = 0.0f;
    float cr12 = 0.0f;
    float cr13 = 0.0f;
    float cr14 = 0.0f;
    float cr21 = 0.0f;
    float cr22 = 0.0f;
    float cr23 = 0.0f;
    float cr24 = 0.0f;
    float cr31 = 0.0f;
    float cr32 = 0.0f;
    float cr33 = 0.0f;
    float cr34 = 0.0f;
    float cr41 = 0.0f;
    float cr42 = 0.0f;
    float cr43 = 0.0f;
    float cr44 = 0.0f;
    for(k = 0; k < l; k += MTHREAD_COUNT) {
      size_t tk;
      as[bi + 0][tj] = 0.0f;
      if(i + 0 < n && k + tj < l) {
        as[bi + 0][tj] = a[l * (i + 0) + k + tj];
      }
      as[bi + 1][tj] = 0.0f;
      if(i + 1 < n && k + tj < l) {
        as[bi + 1][tj] = a[l * (i + 1) + k + tj];
      }
      as[bi + 2][tj] = 0.0f;
      if(i + 2 < n && k + tj < l) {
        as[bi + 2][tj] = a[l * (i + 2) + k + tj];
      }
      as[bi + 3][tj] = 0.0f;
      if(i + 3 < n && k + tj < l) {
        as[bi + 3][tj] = a[l * (i + 3) + k + tj];
      }
      bs[ti][bj + 0] = 0.0f;
      if(j + 0 < m && k + ti < l) {
        bs[ti][bj + 0] = b[l * (j + 0) + k + ti];
      }
      bs[ti][bj + 1] = 0.0f;
      if(j + 1 < m && k + ti < l) {
        bs[ti][bj + 1] = b[l * (j + 1) + k + ti];
      }
      bs[ti][bj + 2] = 0.0f;
      if(j + 2 < m && k + ti < l) {
        bs[ti][bj + 2] = b[l * (j + 2) + k + ti];
      }
      bs[ti][bj + 3] = 0.0f;
      if(j + 3 < m && k + ti < l) {
        bs[ti][bj + 3] = b[l * (j + 3) + k + ti];
      }
      __syncthreads();
      for(tk = 0; tk < MTHREAD_COUNT; tk++) {
        ar1 = as[bi + 0][tk];
        ar2 = as[bi + 1][tk];
        ar3 = as[bi + 2][tk];
        ar4 = as[bi + 3][tk];
        br1 = bs[tk][bj + 0];
        br2 = bs[tk][bj + 1];
        br3 = bs[tk][bj + 2];
        br4 = bs[tk][bj + 3];
        cr11 += ar1 * br1;
        cr12 += ar1 * br2;
        cr13 += ar1 * br3;
        cr14 += ar1 * br4;
        cr21 += ar2 * br1;
        cr22 += ar2 * br2;
        cr23 += ar2 * br3;
        cr24 += ar2 * br4;
        cr31 += ar3 * br1;
        cr32 += ar3 * br2;
        cr33 += ar3 * br3;
        cr34 += ar3 * br4;
        cr41 += ar4 * br1;
        cr42 += ar4 * br2;
        cr43 += ar4 * br3;
        cr44 += ar4 * br4;
      }
      __syncthreads();
    }
    if(i + 0 < n && j + 0 < m) {
      c[m * (i + 0) + j + 0] = cr11;
    }
    if(i + 0 < n && j + 1 < m) {
      c[m * (i + 0) + j + 1] = cr12;
    }
    if(i + 0 < n && j + 2 < m) {
      c[m * (i + 0) + j + 2] = cr13;
    }
    if(i + 0 < n && j + 3 < m) {
      c[m * (i + 0) + j + 3] = cr14;
    }
    if(i + 1 < n && j + 0 < m) {
      c[m * (i + 1) + j + 0] = cr21;
    }
    if(i + 1 < n && j + 1 < m) {
      c[m * (i + 1) + j + 1] = cr22;
    }
    if(i + 1 < n && j + 2 < m) {
      c[m * (i + 1) + j + 2] = cr23;
    }
    if(i + 1 < n && j + 3 < m) {
      c[m * (i + 1) + j + 3] = cr24;
    }
    if(i + 2 < n && j + 0 < m) {
      c[m * (i + 2) + j + 0] = cr31;
    }
    if(i + 2 < n && j + 1 < m) {
      c[m * (i + 2) + j + 1] = cr32;
    }
    if(i + 2 < n && j + 2 < m) {
      c[m * (i + 2) + j + 2] = cr33;
    }
    if(i + 2 < n && j + 3 < m) {
      c[m * (i + 2) + j + 3] = cr34;
    }
    if(i + 3 < n && j + 0 < m) {
      c[m * (i + 3) + j + 0] = cr41;
    }
    if(i + 3 < n && j + 1 < m) {
      c[m * (i + 3) + j + 1] = cr42;
    }
    if(i + 3 < n && j + 2 < m) {
      c[m * (i + 3) + j + 2] = cr43;
    }
    if(i + 3 < n && j + 3 < m) {
      c[m * (i + 3) + j + 3] = cr44;
    }
  }

  __global__ void mul_at_bt(const float *a, const float *b, float *c, size_t n, size_t m, size_t l)
  {
    __shared__ float as[MTILE_WIDTH][MTHREAD_COUNT];
    __shared__ float bs[MTHREAD_COUNT][MTILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x << 2;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y << 2;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    size_t bi = ti << 2;
    size_t bj = tj << 2;
    float ar1;
    float ar2;
    float ar3;
    float ar4;
    float br1;
    float br2;
    float br3;
    float br4;
    float cr11 = 0.0f;
    float cr12 = 0.0f;
    float cr13 = 0.0f;
    float cr14 = 0.0f;
    float cr21 = 0.0f;
    float cr22 = 0.0f;
    float cr23 = 0.0f;
    float cr24 = 0.0f;
    float cr31 = 0.0f;
    float cr32 = 0.0f;
    float cr33 = 0.0f;
    float cr34 = 0.0f;
    float cr41 = 0.0f;
    float cr42 = 0.0f;
    float cr43 = 0.0f;
    float cr44 = 0.0f;
    for(k = 0; k < l; k += MTHREAD_COUNT) {
      size_t tk;
      as[bi + 0][tj] = 0.0f;
      if(i + 0 < n && k + tj < l) {
        as[bi + 0][tj] = a[n * (k + tj) + i + 0];
      }
      as[bi + 1][tj] = 0.0f;
      if(i + 1 < n && k + tj < l) {
        as[bi + 1][tj] = a[n * (k + tj) + i + 1];
      }
      as[bi + 2][tj] = 0.0f;
      if(i + 2 < n && k + tj < l) {
        as[bi + 2][tj] = a[n * (k + tj) + i + 2];
      }
      as[bi + 3][tj] = 0.0f;
      if(i + 3 < n && k + tj < l) {
        as[bi + 3][tj] = a[n * (k + tj) + i + 3];
      }
      bs[ti][bj + 0] = 0.0f;
      if(j + 0 < m && k + ti < l) {
        bs[ti][bj + 0] = b[l * (j + 0) + k + ti];
      }
      bs[ti][bj + 1] = 0.0f;
      if(j + 1 < m && k + ti < l) {
        bs[ti][bj + 1] = b[l * (j + 1) + k + ti];
      }
      bs[ti][bj + 2] = 0.0f;
      if(j + 2 < m && k + ti < l) {
        bs[ti][bj + 2] = b[l * (j + 2) + k + ti];
      }
      bs[ti][bj + 3] = 0.0f;
      if(j + 3 < m && k + ti < l) {
        bs[ti][bj + 3] = b[l * (j + 3) + k + ti];
      }
      __syncthreads();
      for(tk = 0; tk < MTHREAD_COUNT; tk++) {
        ar1 = as[bi + 0][tk];
        ar2 = as[bi + 1][tk];
        ar3 = as[bi + 2][tk];
        ar4 = as[bi + 3][tk];
        br1 = bs[tk][bj + 0];
        br2 = bs[tk][bj + 1];
        br3 = bs[tk][bj + 2];
        br4 = bs[tk][bj + 3];
        cr11 += ar1 * br1;
        cr12 += ar1 * br2;
        cr13 += ar1 * br3;
        cr14 += ar1 * br4;
        cr21 += ar2 * br1;
        cr22 += ar2 * br2;
        cr23 += ar2 * br3;
        cr24 += ar2 * br4;
        cr31 += ar3 * br1;
        cr32 += ar3 * br2;
        cr33 += ar3 * br3;
        cr34 += ar3 * br4;
        cr41 += ar4 * br1;
        cr42 += ar4 * br2;
        cr43 += ar4 * br3;
        cr44 += ar4 * br4;
      }
      __syncthreads();
    }
    if(i + 0 < n && j + 0 < m) {
      c[m * (i + 0) + j + 0] = cr11;
    }
    if(i + 0 < n && j + 1 < m) {
      c[m * (i + 0) + j + 1] = cr12;
    }
    if(i + 0 < n && j + 2 < m) {
      c[m * (i + 0) + j + 2] = cr13;
    }
    if(i + 0 < n && j + 3 < m) {
      c[m * (i + 0) + j + 3] = cr14;
    }
    if(i + 1 < n && j + 0 < m) {
      c[m * (i + 1) + j + 0] = cr21;
    }
    if(i + 1 < n && j + 1 < m) {
      c[m * (i + 1) + j + 1] = cr22;
    }
    if(i + 1 < n && j + 2 < m) {
      c[m * (i + 1) + j + 2] = cr23;
    }
    if(i + 1 < n && j + 3 < m) {
      c[m * (i + 1) + j + 3] = cr24;
    }
    if(i + 2 < n && j + 0 < m) {
      c[m * (i + 2) + j + 0] = cr31;
    }
    if(i + 2 < n && j + 1 < m) {
      c[m * (i + 2) + j + 1] = cr32;
    }
    if(i + 2 < n && j + 2 < m) {
      c[m * (i + 2) + j + 2] = cr33;
    }
    if(i + 2 < n && j + 3 < m) {
      c[m * (i + 2) + j + 3] = cr34;
    }
    if(i + 3 < n && j + 0 < m) {
      c[m * (i + 3) + j + 0] = cr41;
    }
    if(i + 3 < n && j + 1 < m) {
      c[m * (i + 3) + j + 1] = cr42;
    }
    if(i + 3 < n && j + 2 < m) {
      c[m * (i + 3) + j + 2] = cr43;
    }
    if(i + 3 < n && j + 3 < m) {
      c[m * (i + 3) + j + 3] = cr44;
    }
  }

  __global__ void mul_a_b_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] * b[m * i + j];
    }
  }

  __global__ void mul_at_b_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] * b[m * i + j];
    }
  }

  __global__ void mul_a_bt_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] * b[n * j + i];
    }
  }

  __global__ void mul_at_bt_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] * b[n * j + i];
    }
  }

  __global__ void div_a_b_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] / b[m * i + j];
    }
  }

  __global__ void div_at_b_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] / b[m * i + j];
    }
  }

  __global__ void div_a_bt_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] / b[n * j + i];
    }
  }

  __global__ void div_at_bt_for_elems(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] / b[n * j + i];
    }
  }

  __global__ void add_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] + b;
    }
  }

  __global__ void add_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] + b;
    }
  }

  __global__ void sub_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] - b;
    }
  }

  __global__ void sub_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] - b;
    }
  }

  __global__ void rsub_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = b - a[m * i + j];
    }
  }

  __global__ void rsub_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = b - a[n * j + i];
    }
  }

  __global__ void mul_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] * b;
    }
  }

  __global__ void mul_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] * b;
    }
  }

  __global__ void div_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] / b;
    }
  }

  __global__ void div_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[n * j + i] / b;
    }
  }

  __global__ void rdiv_a_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = b / a[m * i + j];
    }
  }

  __global__ void rdiv_at_b_for_scalar(const float *a, float b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = b / a[n * j + i];
    }
  }

  __global__ void sigmoid_a(const float *a, float *b, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      b[m * i + j] = 1.0f / (1.0f + expf(-a[m * i + j]));
    }
  }

  __global__ void sigmoid_at(const float *a, float *b, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      b[m * i + j] = 1.0f / (1.0f + expf(-a[n * j + i]));
    }
  }

  __global__ void tanh_a(const float *a, float *b, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      b[m * i + j] = tanhf(a[m * i + j]);
    }
  }

  __global__ void tanh_at(const float *a, float *b, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      b[m * i + j] = tanhf(a[n * j + i]);
    }
  }

  __global__ void softmax_a(const float *a, float *b, size_t n, size_t m)
  {
    __shared__ float es[TILE_WIDTH][TILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    float sum = 0.0f;
    for(k = 0; k < n; k += TILE_WIDTH) {
      size_t tk;
      es[ti][tj] = 0.0f;
      if(j < m && k + ti < n) {
        es[ti][tj] = exp(a[m * (k + ti) + j]);
      }
      __syncthreads();
      for(tk = 0; tk < TILE_WIDTH; tk++) {
        sum += es[tk][tj];
      }
      __syncthreads();
    }
    if(i < n && j < m) {
      b[m * i + j] = exp(a[m * i + j]) / sum;
    }
  }

  __global__ void softmax_at(const float *a, float *b, size_t n, size_t m)
  {
    __shared__ float es[TILE_WIDTH][TILE_WIDTH];
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    size_t k;
    size_t ti = threadIdx.x;
    size_t tj = threadIdx.y;
    float sum = 0.0f;
    for(k = 0; k < n; k += TILE_WIDTH) {
      size_t tk;
      es[ti][tj] = 0.0f;
      if(j < m && k + ti < n) {
        es[ti][tj] = exp(a[n * j + k + ti]);
      }
      __syncthreads();
      for(tk = 0; tk < TILE_WIDTH; tk++) {
        sum += es[tk][tj];
      }
      __syncthreads();
    }
    if(i < n && j < m) {
      b[m * i + j] = exp(a[n * j + i]) / sum;
    }
  }
}

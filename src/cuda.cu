
#include <hip/hip_runtime.h>
//
// Copyright (c) 2025 Łukasz Szpakowski
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.
//
extern "C" {
  __global__ void add_a_b(const float *a, const float *b, float *c, size_t n, size_t m)
  {
    size_t i = ((size_t) blockDim.x) * blockIdx.x + threadIdx.x;
    size_t j = ((size_t) blockDim.y) * blockIdx.y + threadIdx.y;
    if(i < n && j < m) {
      c[m * i + j] = a[m * i + j] + b[m * i + j];
    }
  }
}
